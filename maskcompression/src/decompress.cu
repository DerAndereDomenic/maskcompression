#include "hip/hip_runtime.h"
#include "decompress.h"

#include <ATen/cuda/ApplyGridUtils.cuh>
#include <c10/cuda/CUDAGuard.h>
#include <iostream>

namespace maskcompression
{

namespace detail
{

template<typename T>
inline __device__ uint32_t
binary_search(const torch::PackedTensorAccessor32<T, 1, torch::RestrictPtrTraits> sorted_array, T value)
{
    // Find first element in sorted_array that is larger than value.
    uint32_t left  = 0;
    uint32_t right = sorted_array.size(0) - 1;
    while(left < right)
    {
        uint32_t mid = (left + right) / 2;
        if(sorted_array[mid] < value)
            left = mid + 1;
        else
            right = mid;
    }
    return left;
}

__global__ void decompressImage(const torch::PackedTensorAccessor32<int32_t, 1, torch::RestrictPtrTraits> cumsum,
                                const uint32_t width,
                                const uint32_t height,
                                torch::PackedTensorAccessor32<float, 2, torch::RestrictPtrTraits> output)
{
    auto id = static_cast<int64_t>(blockIdx.x) * static_cast<int64_t>(blockDim.x) + static_cast<int64_t>(threadIdx.x);
    auto num_threads = static_cast<int64_t>(gridDim.x) * static_cast<int64_t>(blockDim.x);
    for(int32_t tid = id; tid < width * height; tid += num_threads)
    {
        int pixel_x = tid % width;
        int pixel_y = tid / width;

        uint32_t bin_index = binary_search(cumsum, tid + 1);

        output[pixel_y][pixel_x] = bin_index % 2 == 0 ? 0.0f : 1.0f;
    }
}
}    // namespace detail

torch::Tensor decompress(const torch::Tensor& compressed, at::IntArrayRef& resolution)
{
    torch::Tensor output =
        torch::zeros(resolution, torch::TensorOptions {}.dtype(torch::kFloat32).device(torch::kCUDA));
    auto cumsum = torch::cumsum(compressed, 0).to(torch::kInt32);    // TODO: copy

    auto device = output.device();

    at::cuda::CUDAGuard device_guard {device};
    const auto stream = at::cuda::getCurrentCUDAStream();

    const int threads_per_block = 128;
    dim3 grid;
    at::cuda::getApplyGrid(resolution[0] * resolution[1], grid, device.index(), threads_per_block);
    dim3 threads = at::cuda::getApplyBlock(threads_per_block);

    detail::decompressImage<<<grid, threads, 0, stream>>>(
        cumsum.packed_accessor32<int32_t, 1, torch::RestrictPtrTraits>(),
        resolution[1],
        resolution[0],
        output.packed_accessor32<float, 2, torch::RestrictPtrTraits>());

    AT_CUDA_CHECK(hipGetLastError());
    AT_CUDA_CHECK(hipStreamSynchronize(stream));

    return output;
}
}    // namespace maskcompression