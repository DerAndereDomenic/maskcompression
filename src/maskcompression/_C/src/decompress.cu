#include "hip/hip_runtime.h"
#include <maskcompression/decompress.h>

#include <ATen/cuda/ApplyGridUtils.cuh>
#include <c10/cuda/CUDAGuard.h>
#include <iostream>

namespace maskcompression
{

namespace detail
{

inline __device__ uint32_t
binary_search(const torch::PackedTensorAccessor32<int32_t, 1, torch::RestrictPtrTraits>& sorted_array, int32_t value)
{
    // Find first element in sorted_array that is larger than value.
    uint32_t left  = 1;    // Start at 1 because index 0 encodes if the mask starts with 0 or 1
    uint32_t right = sorted_array.size(0) - 1;
    while(left < right)
    {
        uint32_t mid = (left + right) / 2;
        if(sorted_array[mid] < value)
            left = mid + 1;
        else
            right = mid;
    }
    return left;
}

__global__ void decompressImage(const torch::PackedTensorAccessor32<int32_t, 1, torch::RestrictPtrTraits> cumsum,
                                const uint32_t batch_id,
                                const uint32_t width,
                                const uint32_t height,
                                torch::PackedTensorAccessor32<float, 3, torch::RestrictPtrTraits> output)
{
    auto id = static_cast<int64_t>(blockIdx.x) * static_cast<int64_t>(blockDim.x) + static_cast<int64_t>(threadIdx.x);
    auto num_threads = static_cast<int64_t>(gridDim.x) * static_cast<int64_t>(blockDim.x);
    for(int32_t tid = id; tid < width * height; tid += num_threads)
    {
        int pixel_x = tid % width;
        int pixel_y = tid / width;

        uint32_t bin_index =
            binary_search(cumsum, tid + 1) - 1;    // -1 because index 0 encodes if the mask starts with 0 or 1

        int32_t leading_one = cumsum[0];

        output[batch_id][pixel_y][pixel_x] = ((bin_index + leading_one) & 1) ? 1.0f : 0.0f;
    }
}
}    // namespace detail

torch::Tensor decompress(const std::vector<torch::Tensor>& compressed, at::IntArrayRef& resolution)
{
    int batch_size       = compressed.size();
    torch::Tensor output = torch::zeros({batch_size, resolution[0], resolution[1]},
                                        torch::TensorOptions {}.dtype(torch::kFloat32).device(torch::kCUDA));

    auto device = output.device();

    at::cuda::CUDAGuard device_guard {device};
    const auto stream = at::cuda::getCurrentCUDAStream();

    const int threads_per_block = 128;
    dim3 grid;
    at::cuda::getApplyGrid(resolution[0] * resolution[1], grid, device.index(), threads_per_block);
    dim3 threads = at::cuda::getApplyBlock(threads_per_block);

    for(int batch_id = 0; batch_id < batch_size; ++batch_id)
    {
        auto cumsum = compressed[batch_id];
        detail::decompressImage<<<grid, threads, 0, stream>>>(
            cumsum.packed_accessor32<int32_t, 1, torch::RestrictPtrTraits>(),
            batch_id,
            resolution[1],
            resolution[0],
            output.packed_accessor32<float, 3, torch::RestrictPtrTraits>());
    }

    AT_CUDA_CHECK(hipGetLastError());
    AT_CUDA_CHECK(hipStreamSynchronize(stream));

    return output;
}
}    // namespace maskcompression