#include "hip/hip_runtime.h"
#include <maskcompression/decompress.h>

#include <ATen/cuda/ApplyGridUtils.cuh>
#include <c10/cuda/CUDAGuard.h>
#include <iostream>

namespace maskcompression
{

namespace detail
{

inline __device__ uint32_t
binary_search(const torch::PackedTensorAccessor32<int32_t, 1, torch::RestrictPtrTraits>& sorted_array, int32_t value)
{
    // Find first element in sorted_array that is larger than value.
    uint32_t left  = 1;    // Start at 1 because index 0 encodes if the mask starts with 0 or 1
    uint32_t right = sorted_array.size(0) - 1;
    while(left < right)
    {
        uint32_t mid = (left + right) / 2;
        if(sorted_array[mid] < value)
            left = mid + 1;
        else
            right = mid;
    }
    return left;
}

__global__ void decompressImage(const torch::PackedTensorAccessor32<int32_t, 1, torch::RestrictPtrTraits> cumsum,
                                const uint32_t batch_id,
                                const uint32_t width,
                                const uint32_t height,
                                const bool vertical_flip,
                                torch::PackedTensorAccessor32<uint8_t, 3, torch::RestrictPtrTraits> output)
{
    auto id = static_cast<int64_t>(blockIdx.x) * static_cast<int64_t>(blockDim.x) + static_cast<int64_t>(threadIdx.x);
    auto num_threads = static_cast<int64_t>(gridDim.x) * static_cast<int64_t>(blockDim.x);
    for(int32_t tid = id; tid < width * height; tid += num_threads)
    {
        int pixel_x = tid % width;
        int pixel_y = vertical_flip ? height - tid / width - 1 : tid / width;

        uint32_t bin_index =
            binary_search(cumsum, tid + 1) - 1;    // -1 because index 0 encodes if the mask starts with 0 or 1

        int32_t leading_one = cumsum[0];

        output[batch_id][pixel_y][pixel_x] = ((bin_index + leading_one) & 1) ? 1 : 0;
    }
}
}    // namespace detail

torch::Tensor
decompress(const std::vector<torch::Tensor>& compressed, const std::array<int, 2>& resolution, const bool vertical_flip)
{
    if(resolution[0] <= 0 || resolution[1] <= 1)
    {
        std::stringstream ss;
        ss << "Got invalid input resolution: " << resolution[0] << ", " << resolution[1];
        throw std::runtime_error(ss.str());
    }

    int batch_size       = compressed.size();
    torch::Tensor output = torch::zeros({batch_size, resolution[0], resolution[1]},
                                        torch::TensorOptions {}.dtype(torch::kUInt8).device(torch::kCUDA));

    auto device = output.device();

    at::cuda::CUDAGuard device_guard {device};
    const auto stream = at::cuda::getCurrentCUDAStream();

    const int threads_per_block = 128;
    dim3 grid;
    at::cuda::getApplyGrid(resolution[0] * resolution[1], grid, device.index(), threads_per_block);
    dim3 threads = at::cuda::getApplyBlock(threads_per_block);

    for(int batch_id = 0; batch_id < batch_size; ++batch_id)
    {
        auto cumsum = compressed[batch_id];

        if(!cumsum.device().is_cuda())
        {
            throw std::runtime_error("maskcompression only works for CUDA capable devices");
        }

        if(cumsum.device() != device)
        {
            std::stringstream ss;
            ss << "All compressed masks have to be on the same device. Got: " << cumsum.device() << " and " << device;
            throw std::runtime_error(ss.str());
        }

        if(cumsum.scalar_type() != torch::kInt32)
        {
            std::stringstream ss;
            ss << "Got unexpected datatype for maskcompression. Expected torch.int32, got " << cumsum.scalar_type();
            throw std::runtime_error(ss.str());
        }

        detail::decompressImage<<<grid, threads, 0, stream>>>(
            cumsum.packed_accessor32<int32_t, 1, torch::RestrictPtrTraits>(),
            batch_id,
            resolution[1],
            resolution[0],
            vertical_flip,
            output.packed_accessor32<uint8_t, 3, torch::RestrictPtrTraits>());
    }

    AT_CUDA_CHECK(hipGetLastError());
    AT_CUDA_CHECK(hipStreamSynchronize(stream));

    return output;
}
}    // namespace maskcompression